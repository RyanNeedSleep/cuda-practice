#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "sq_matmul.cuh"
#include <iostream>
#include <cassert>

#define TILE_WIDTH 32

__global__
void sq_matmul_uncoalesced(const float* A, const float* B, float* C, int N){
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N && j < N){
        float value = 0.0f;
        for (int k = 0; k < N; ++k){
            value += A[i * N + k] * B[k * N + j];
        }
        C[i * N + j] = value;
    }
    return;
}

__global__
void sq_matmul_coalesced(const float* A, const float* B, float* C, int N){
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N && j < N){
        float value = 0.0f;
        for (int k = 0; k < N; ++k){
            value += A[i * N + k] * B[k * N + j];
        }
        C[i * N + j] = value;
    }
    return;
}

__global__
void sq_matmul_sharedmem(const float* A, const float* B, float* C, int N){
    assert(TILE_WIDTH == blockDim.x);
    assert(TILE_WIDTH == blockDim.y);

    assert(N % TILE_WIDTH == 0);



    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N && j < N){

        __shared__ float As[TILE_WIDTH][TILE_WIDTH];
        __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

        float value = 0.0f;
        for (int phase = 0; phase < N/TILE_WIDTH; ++phase){
            As[threadIdx.y][threadIdx.x] = A[i * N + phase * TILE_WIDTH + threadIdx.x];
            Bs[threadIdx.y][threadIdx.x] = B[(phase * TILE_WIDTH + threadIdx.y) * N + j];
            __syncthreads();

            for (int k = 0; k < TILE_WIDTH; ++k){
                value += As[threadIdx.y][k] * Bs[k][threadIdx.x];
            }
            __syncthreads();
        }
        C[i * N + j] = value;
    }
    return;
}


void launch_sq_matmul(const float* A, const float* B, float* C, int N, const std::string& version){
    dim3 block(32, 32);
    dim3 grid((N + block.x - 1) / block.x, (N + block.y - 1) / block.y);
    if (version == "uncoalesced"){
        sq_matmul_uncoalesced<<<grid, block>>>(A, B, C, N);
    }else if (version == "coalesced"){
        sq_matmul_coalesced<<<grid, block>>>(A, B, C, N);
    }else if (version == "shared"){
        sq_matmul_sharedmem<<<grid, block>>>(A, B, C, N);
    }else{
        std::cout << "please select type of operation" << std::endl;
    }
}
